#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define BW 16 // Block Width
#define BH 32 // Block Height
#define COUNT 0


// Kernel Function handles first nested for loop
__global__ void kernelBlur(int *d_Rnew, int *d_Gnew, int *d_Bnew, int *d_R, int *d_G, int *d_B, int rowsize, int colsize) {
	// Set-up
	int row = blockIdx.y*blockDim.y + threadIdx.y;
    	int col = blockIdx.x*blockDim.x + threadIdx.x;
	// Run Some Calculations
	if (col < colsize && row < rowsize) {
		if (row != 0 && row != (rowsize-1) && col != 0 && col != (colsize-1)) {
			d_Rnew[row * colsize + col] = (d_R[(row + 1) * colsize + col]+d_R[(row - 1) * colsize + col]+d_R[row * colsize + (col + 1)]+d_R[row * colsize + (col - 1)])/4;
			d_Gnew[row * colsize + col] = (d_G[(row + 1) * colsize + col]+d_G[(row - 1) * colsize + col]+d_G[row * colsize + (col + 1)]+d_G[row * colsize + (col - 1)])/4;
			d_Bnew[row * colsize + col] = (d_B[(row + 1) * colsize + col]+d_B[(row - 1) * colsize + col]+d_B[row * colsize + (col + 1)]+d_B[row * colsize + (col - 1)])/4;
		}
		else if (row == 0 && col != 0 && col != (colsize-1)){
			d_Rnew[row * colsize + col] = (d_R[(row + 1)  * colsize + col]+d_R[row * colsize + (col + 1)]+d_R[row * colsize + (col - 1)])/3;
			d_Gnew[row * colsize + col] = (d_G[(row + 1)  * colsize + col]+d_G[row * colsize + (col + 1)]+d_G[row * colsize + (col - 1)])/3;
			d_Bnew[row * colsize + col] = (d_B[(row + 1)  * colsize + col]+d_B[row * colsize + (col + 1)]+d_B[row * colsize + (col - 1)])/3;
		}
		else if (row == (rowsize-1) && col != 0 && col != (colsize-1)){
			d_Rnew[row * colsize + col] = (d_R[(row - 1) * colsize + col]+d_R[row * colsize + (col + 1)]+d_R[row * colsize + (col - 1)])/3;
			d_Gnew[row * colsize + col] = (d_G[(row - 1) * colsize + col]+d_G[row * colsize + (col + 1)]+d_G[row * colsize + (col - 1)])/3;
			d_Bnew[row * colsize + col] = (d_B[(row - 1) * colsize + col]+d_B[row * colsize + (col + 1)]+d_B[row * colsize + (col - 1)])/3;
		}
		else if (col == 0 && row != 0 && row != (rowsize-1)){
			d_Rnew[row * colsize + col] = (d_R[(row + 1) * colsize + col]+d_R[(row - 1) * colsize + col]+d_R[row * colsize + (col + 1)])/3;
			d_Gnew[row * colsize + col] = (d_G[(row + 1) * colsize + col]+d_G[(row - 1) * colsize + col]+d_G[row * colsize + (col + 1)])/3;
			d_Bnew[row * colsize + col] = (d_B[(row + 1) * colsize + col]+d_B[(row - 1) * colsize + col]+d_B[row * colsize + (col + 1)])/3;
		}
		else if (col == (colsize-1) && row != 0 && row != (rowsize-1)){
			d_Rnew[row * colsize + col] = (d_R[(row + 1) * colsize + col]+d_R[(row - 1) * colsize + col]+d_R[row * colsize + (col + 1)])/3;
			d_Gnew[row * colsize + col] = (d_G[(row + 1) * colsize + col]+d_G[(row - 1) * colsize + col]+d_G[row * colsize + (col + 1)])/3;
			d_Bnew[row * colsize + col] = (d_B[(row + 1) * colsize + col]+d_B[(row - 1) * colsize + col]+d_B[row * colsize + (col + 1)])/3;
		}
		else if (row==0 &&col==0){
			d_Rnew[row * colsize + col] = (d_R[row * colsize + (col + 1)]+d_R[(row + 1) * colsize + col])/2;
			d_Gnew[row * colsize + col] = (d_G[row * colsize + (col + 1)]+d_G[(row + 1) * colsize + col])/2;
			d_Bnew[row * colsize + col] = (d_B[row * colsize + (col + 1)]+d_B[(row + 1) * colsize + col])/2;
		}
		else if (row==0 &&col==(colsize-1)){
			d_Rnew[row * colsize + col] = (d_R[row * colsize + (col - 1)]+d_R[(row + 1) * colsize + col])/2;
			d_Gnew[row * colsize + col] = (d_G[row * colsize + (col - 1)]+d_G[(row + 1) * colsize + col])/2;
			d_Bnew[row * colsize + col] = (d_B[row * colsize + (col - 1)]+d_B[(row + 1) * colsize + col])/2;
		}
		else if (row==(rowsize-1) &&col==0){
			d_Rnew[row * colsize + col] = (d_R[row * colsize + (col + 1)]+d_R[(row - 1) * colsize + col])/2;
			d_Gnew[row * colsize + col] = (d_G[row * colsize + (col + 1)]+d_G[(row - 1) * colsize + col])/2;
			d_Bnew[row * colsize + col] = (d_B[row * colsize + (col + 1)]+d_B[(row - 1) * colsize + col])/2;
		}
		else if (row==(rowsize-1) &&col==(colsize-1)){
			d_Rnew[row * colsize + col] = (d_R[row * colsize + (col - 1)]+d_R[(row - 1) * colsize + col])/2;
			d_Gnew[row * colsize + col] = (d_G[row * colsize + (col - 1)]+d_G[(row - 1) * colsize + col])/2;
			d_Bnew[row * colsize + col] = (d_B[row * colsize + (col - 1)]+d_B[(row - 1) * colsize + col])/2;
		}
	}
}

// Kernel Function handles second nested for loop updates RGB values to new calculated values
__global__ void kernelCopy(int *d_Rnew, int *d_Gnew, int *d_Bnew, int *d_R, int *d_G, int *d_B, int rowsize, int colsize) {
	// Set-up
	int row = blockIdx.y*blockDim.y+threadIdx.y;
    	int col = blockIdx.x*blockDim.x+threadIdx.x;
	if (col < colsize && row < rowsize) {
		d_R[row * colsize + col] = d_Rnew[row * colsize + col];
		d_G[row * colsize + col] = d_Gnew[row * colsize + col];
		d_B[row * colsize + col] = d_Bnew[row * colsize + col];

	}
}

void performBlurs(int *h_R, int *h_G, int *h_B, int *h_Rnew, int *h_Gnew, int *h_Bnew, int rowsize, int colsize, int nblurs) {
	// Assign Memory on GPU
	// Step 1 Assign Memory on GPU
	int k;
	int sizei = sizeof(int)*rowsize*colsize;
	int *d_R, *d_G, *d_B, *d_Rnew, *d_Gnew, *d_Bnew;

	struct timeval tim;
	gettimeofday(&tim, NULL);
	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);

	hipMalloc((void **)&d_R,sizei);
	hipMalloc((void **)&d_G,sizei);
	hipMalloc((void **)&d_B,sizei);
	hipMalloc((void **)&d_Rnew,sizei);
	hipMalloc((void **)&d_Gnew,sizei);
	hipMalloc((void **)&d_Bnew,sizei);

	gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("Assigning Memory to GPU > %.6lf seconds elapsed\n", t2-t1);

	// Transfer to Device
	gettimeofday(&tim, NULL);
	t1=tim.tv_sec+(tim.tv_usec/1000000.0);

	hipMemcpy(d_R, h_R, sizei, hipMemcpyHostToDevice);
	hipMemcpy(d_G, h_G, sizei, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizei, hipMemcpyHostToDevice);
	hipMemcpy(d_Rnew, h_Rnew, sizei, hipMemcpyHostToDevice);
	hipMemcpy(d_Gnew, h_Gnew, sizei, hipMemcpyHostToDevice);
	hipMemcpy(d_Bnew, h_Bnew, sizei, hipMemcpyHostToDevice);

	t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("Transferring from host to device memory > %.6lf seconds elapsed\n", t2-t1);

	// Set up Blocks
	dim3 dimGrid(ceil(colsize/(float)BW), ceil(rowsize/(float)BH), 1);
	dim3 dimBlock(BW,BH);

	nblurs = 10; // Modify as Needed
	gettimeofday(&tim, NULL);
	t1=tim.tv_sec+(tim.tv_usec/1000000.0);
	for (k = 0; k < nblurs; ++k) {
		kernelBlur<<<dimGrid, dimBlock>>>(d_Rnew, d_Gnew, d_Bnew, d_R, d_G, d_B, rowsize, colsize);
		kernelCopy<<<dimGrid, dimBlock>>>(d_Rnew, d_Gnew, d_Bnew, d_R, d_G, d_B, rowsize, colsize);
	}
	t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("Blurring Operation > %.6lf seconds elapsed\n", t2-t1);

	// Step 4 output copied from GPU to Host get the RGB values
	hipMemcpy(h_R, d_R, sizei, hipMemcpyDeviceToHost);
	hipMemcpy(h_G, d_G, sizei, hipMemcpyDeviceToHost);
	hipMemcpy(h_B, d_B, sizei, hipMemcpyDeviceToHost);

	// Step 5 Free Memory
	hipFree(d_R); hipFree(d_G); hipFree(d_B); hipFree(d_Rnew); hipFree(d_Gnew); hipFree(d_Bnew);
}

int main (int argc, const char * argv[]) {
	// Assignment of initial Variables
	static int const maxlen = 200, rowsize = 521, colsize = 428, linelen = 12;
	static char str[200], lines[5][200];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;

	// Define Host Arrays
	int *h_R, *h_G, *h_B;
	int *h_Rnew, *h_Gnew, *h_Bnew;
	int size = sizeof(int) * rowsize * colsize;
	h_R = (int *)malloc(size);
	h_G = (int *)malloc(size);
	h_B = (int *)malloc(size);
	h_Rnew = (int *)malloc(size);
	h_Gnew = (int *)malloc(size);
	h_Bnew = (int *)malloc(size);

	// Allocate Overall Size of ROw

	int row = 0, col = 0, nblurs = 0, lineno=0, k;

	// Read input file
	struct timeval tim;
	gettimeofday(&tim, NULL);
	double t1=tim.tv_sec+(tim.tv_usec/1000000.0);

	fp = fopen("sample.ps", "r");
	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}
		else{
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);

				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize) {
					h_R[row * colsize + col] = h1;
					h_G[row * colsize + col] = h2;
					h_B[row * colsize + col] = h3;
				}
				col++;
			}
		}
	}
	fclose(fp);
	gettimeofday(&tim, NULL);
	double t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("Reading Input File > %.6lf seconds elapsed\n", t2-t1);

	// Run Code
	performBlurs(h_R, h_G, h_B, h_Rnew, h_Gnew, h_Bnew, rowsize, colsize, nblurs);

	gettimeofday(&tim, NULL);
	t1=tim.tv_sec+(tim.tv_usec/1000000.0);
	fout= fopen("sampleBlurCU.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",h_R[row * colsize + col],h_G[row * colsize + col],h_B[row * colsize + col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	gettimeofday(&tim, NULL);
	t2=tim.tv_sec+(tim.tv_usec/1000000.0);
	printf("Outputting File > %.6lf seconds elapsed\n", t2-t1);
	fclose(fout);
    return 0;
}
